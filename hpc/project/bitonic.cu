#include "hip/hip_runtime.h"
#include <stdio.h>

// The original array, stored in global memory. The final result will
// eventually overwrite the original and be stored here.
__device__ float* array;
 
__global__ __forceinline__ void bitonicSort(float* a, float* b);
__global__ __forceinline__ void bitonicBuild(float* a, float* b);
void bitonicBuildRunner(float* a, int size);
void bitonicSortRunner(float* a, int size);
 
int main(int argc, char **argv) {
	// Input in array...
    float* array;
	
	// BEGIN
    int n = atoi(argv[1]), i,s;
	
	
	array = (float*) malloc(n * sizeof(int));
	for(i = 0; i < n; i++) {
		array[i] = rand();
	}
	// END
    
	// Size of array;
    int size;
	
	// Transfer flow of control to device
    bitonicBuildRunner(array, size);
	bitonicSortRunner(array, size);
}

void bitonicSortRunner(float* a, int size) {
	// Copy over memory
    float* array;
	int mem = sizeof(float) * size;
	hipMalloc(array, mem);
	hipMemcpy(array, a, mem, hipMemcpyHostToDevice);
	
	int blocks = 1;
	while(blocks != size / 2) {
		// Execution config
        dim3 numBlocks = blocks;
		dim3 threadsPerBlock = size / blocks / 2;
		
		bitonicSort<<<numBlocks, threadsPerBlock>>>(array, size / blocks);
		size *= 2;
	}
}

void bitonicBuildRunner(float* a, int size) {
	int blocks = size / 2;
	while(blocks != 1) {
		int i = blocks, blockSize = size * (1 - 1 / blocks);
		while(i != 1) {
			dim3 numBlocks = i, threadsPerBlock = blockSize;
			for(j = 0; j < size; j += blockSize, a++) {
				bitonicBuild<<<numBlocks, threadsPerBlock>>>(a, blockSize, i);
			}
			i /= 2;
		}
		blocks /= 2;
	}
}

/**
 * Applies the bitonic sorting algorithm to each thread. It swaps two
 * elements in the two lists if they are out of place.
 */
__global__ __forceinline__ void bitonicSort(float* a, int blockSize) {
	// First we need to figure out what index each thread will access
    int index = threadIdx.x + blockIdx.x * blockSize;
	atomicMin(&a[i + index], 
		atomicMax(&a[i + blockSize + index], a[i + index]));
	__syncthreads();
}

/**
 * Combines two bitonic sequences together to create a new bitonic sequence.
 * @param a Pointer to the start of the bitonic sequence.
 * @param blockSize The size of each sub-array partition.
 * @param t Determines when to switch between ascending and descending.
 */
__global__ __forceinline__ void bitonicBuild(float* a, int blockSize, int t) {
	int index = threadIdx.x + blockIdx.x * blockSize, x = 0, asc = 1;
	float* b = a + index + (blockSize / 2);
	while(x > index) {
		x += t;
		asc++;
	}
	
	if(asc % 2 == 1) {
		atomicMin(&a[index], atomicMax(&b, a[index]));
	}
	else {
		atomicMax(&a[index], atomicMin(&b, a[index]));
	}
}