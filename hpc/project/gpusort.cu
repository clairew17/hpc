#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/sort.h>
#include <thrust/merge.h>

#define NUM_SETS 100000
#define DSIZE 100
typedef int mytype;

// for ascending sorted data
#define cmp(A,B) ((A)<(B))
#define nTPB 512
#define nBLK 128

#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

template <typename T>
__host__ __device__ void smerge(const T * __restrict__  a, const T * __restrict__ b, T * __restrict__ c, const unsigned len_a, const unsigned len_b, const unsigned stride_a = 1, const unsigned stride_b = 1, const unsigned stride_c = 1){

  unsigned len_c = len_a+len_b;
  unsigned nc = 0;
  unsigned na = 0;
  unsigned nb = 0;
  unsigned fa = (len_b == 0);
  unsigned fb = (len_a == 0);
  T nxta = a[0];
  T nxtb = b[0];
  while (nc < len_c){
    if (fa)      {c[stride_c*nc++] = nxta; na++; nxta = a[stride_a*na];}
    else if (fb) {c[stride_c*nc++] = nxtb; nb++; nxtb = b[stride_b*nb];}
    else if (cmp(nxta,nxtb)){
      c[stride_c*nc++] = nxta;
      na++;
      if (na == len_a) fb++;
      else nxta = a[stride_a*na];}
    else {
      c[stride_c*nc++] = nxtb;
      nb++;
      if (nb == len_b) fa++;
      else nxtb = b[stride_b*nb];}}
}



template <typename T>
__global__ void rmtest(const T * __restrict__  a, const T * __restrict__ b, T * __restrict__  c, int num_arr, int len){

  int idx=threadIdx.x+blockDim.x*blockIdx.x;

  while (idx < num_arr){
    int sel=idx*len;
    smerge(a+sel, b+sel, c+(2*sel), len, len);
    idx += blockDim.x*gridDim.x;}
}

template <typename T>
__global__ void cmtest(const T * __restrict__ a, const T * __restrict__ b, T * __restrict__ c, int num_arr, int len, int stride_a, int stride_b, int stride_c){
  int idx=threadIdx.x+blockDim.x*blockIdx.x;
  while (idx < num_arr){
    smerge(a+idx, b+idx, c+idx, len, len, stride_a, stride_b, stride_c);
    idx += blockDim.x*gridDim.x;}
}




template <typename T>
int rmvalidate(T *a, T *b, T *c, int num_arr, int len){

  T *vc = (T *)malloc(2*len*sizeof(T));
  for (int i = 0; i < num_arr; i++){
    thrust::merge(a+(i*len), a+((i+1)*len), b+(i*len), b+((i+1)*len), vc);
#ifndef TIMING
    for (int j = 0; j < len*2; j++)
      if (vc[j] != c[(i*2*len)+j]) {printf("rm mismatch i: %d, j: %d, was: %d, should be: %d\n", i, j, c[(i*2*len)+j], vc[j]); return 0;}
#endif
    }
  return 1;
}

template <typename T>
int cmvalidate(const T *c1, const T *c2, int num_arr, int len){
  for (int i = 0; i < num_arr; i++)
    for (int j = 0; j < 2*len; j++)
      if (c1[i*(2*len)+j] != c2[j*(num_arr)+i]) {printf("cm mismatch i: %d, j: %d, was: %d, should be: %d\n", i, j, c2[j*(num_arr)+i], c1[i*(2*len)+j]); return 0;}
  return 1;
}

int main(){


  mytype *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
  h_a = (mytype *)malloc(DSIZE*NUM_SETS*sizeof(mytype));
  h_b = (mytype *)malloc(DSIZE*NUM_SETS*sizeof(mytype));
  h_c = (mytype *)malloc(DSIZE*NUM_SETS*sizeof(mytype)*2);
  hipMalloc(&d_a, (DSIZE*NUM_SETS+1)*sizeof(mytype));
  hipMalloc(&d_b, (DSIZE*NUM_SETS+1)*sizeof(mytype));
  hipMalloc(&d_c, DSIZE*NUM_SETS*sizeof(mytype)*2);
// test "row-major" storage
  for (int i =0; i<DSIZE*NUM_SETS; i++){
    h_a[i] = rand();
    h_b[i] = rand();}
  thrust::sort(h_a, h_a+DSIZE*NUM_SETS);
  thrust::sort(h_b, h_b+DSIZE*NUM_SETS);
  hipMemcpy(d_a, h_a, DSIZE*NUM_SETS*sizeof(mytype), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, DSIZE*NUM_SETS*sizeof(mytype), hipMemcpyHostToDevice);
  unsigned long gtime = dtime_usec(0);
  rmtest<<<nBLK, nTPB>>>(d_a, d_b, d_c, NUM_SETS, DSIZE);
  hipDeviceSynchronize();
  gtime = dtime_usec(gtime);
  hipMemcpy(h_c, d_c, DSIZE*NUM_SETS*2*sizeof(mytype), hipMemcpyDeviceToHost);
  unsigned long ctime = dtime_usec(0);
  if (!rmvalidate(h_a, h_b, h_c, NUM_SETS, DSIZE)) {printf("fail!\n"); return 1;}
  ctime = dtime_usec(ctime);
  printf("CPU time: %f, GPU RM time: %f\n", ctime/(float)USECPSEC, gtime/(float)USECPSEC);
// test "col-major" storage
  mytype *ch_a, *ch_b, *ch_c;
  ch_a = (mytype *)malloc(DSIZE*NUM_SETS*sizeof(mytype));
  ch_b = (mytype *)malloc(DSIZE*NUM_SETS*sizeof(mytype));
  ch_c = (mytype *)malloc(DSIZE*NUM_SETS*sizeof(mytype));
  for (int i = 0; i < NUM_SETS; i++)
    for (int j = 0; j < DSIZE; j++){
      ch_a[j*NUM_SETS+i] = h_a[i*DSIZE+j];
      ch_b[j*NUM_SETS+i] = h_b[i*DSIZE+j];}
  hipMemcpy(d_a, ch_a, DSIZE*NUM_SETS*sizeof(mytype), hipMemcpyHostToDevice);
  hipMemcpy(d_b, ch_b, DSIZE*NUM_SETS*sizeof(mytype), hipMemcpyHostToDevice);
  gtime = dtime_usec(0);
  cmtest<<<nBLK, nTPB>>>(d_a, d_b, d_c, NUM_SETS, DSIZE, NUM_SETS, NUM_SETS, NUM_SETS );
  hipDeviceSynchronize();
  gtime = dtime_usec(gtime);
  hipMemcpy(ch_c, d_c, DSIZE*NUM_SETS*2*sizeof(mytype), hipMemcpyDeviceToHost);
  if (!cmvalidate(h_c, ch_c, NUM_SETS, DSIZE)) {printf("fail!\n"); return 1;}

  printf("GPU CM time: %f\n", gtime/(float)USECPSEC);
  return 0;
}